#include "hip/hip_runtime.h"
#include "kernel.h"

#include <memory>

__global__ void multiplyKernet(float *a, float *b, float *c)
{
	extern __shared__ float shared_a[SIZE*SIZE];
	extern __shared__ float shared_b[SIZE*SIZE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	shared_a[tx*SIZE + ty] = a[tx*SIZE + ty];
	shared_b[tx*SIZE + ty] = b[tx*SIZE + ty];
	float loc_c = 0.0;
	__syncthreads();
	for (int k = 0; k < SIZE; ++k)
	{
		loc_c += shared_a[tx*k+ty] * shared_b[ty*k+tx];
	}
	c[tx*SIZE + ty] = loc_c;
	__syncthreads();
}


hipError_t cudaCopy(float *dest, float *src, size_t size, hipMemcpyKind copyKind)
{
	hipError_t cudaStatus = hipMemcpy(dest, src, size, copyKind);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed: %s", hipGetErrorString(cudaStatus));
	}
	return cudaStatus;
}

hipError_t cudaAllocate(void **devPtr, size_t size)
{
	hipError_t cudaStatus = hipMalloc(devPtr, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed: %s", hipGetErrorString(cudaStatus));
	}

	return cudaStatus;
}

hipError_t cudaCall(hipError_t(*cudaFunc)(), const char *msg)
{
	hipError_t cudaStatus = cudaFunc();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(cudaStatus));
	}
	return cudaStatus;
}

hipError_t cudaStartTimer(hipEvent_t *start, hipEvent_t *stop)
{
	hipError_t cudaStatus = hipEventCreate(start);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaCEventCreate(start) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	cudaStatus = hipEventCreate(stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventCreate(stop) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	cudaStatus = hipEventRecord(*start, 0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventRecord(start) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	return hipSuccess;
}

hipError_t cudaStopTimer(hipEvent_t *start, hipEvent_t *stop, float & msecTotal)
{
	hipError_t cudaStatus = hipEventRecord(*stop, 0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventRecord(stop) failed !\n");
		return cudaStatus;
	}
	cudaStatus = hipEventSynchronize(*stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventSynchronize failed !\n");
		return cudaStatus;
	}
	msecTotal = 0.0f;
	cudaStatus = hipEventElapsedTime(&msecTotal, *start, *stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventElapsedTime failed !\n");
		return cudaStatus;
	}

	cudaStatus = hipEventDestroy(*start);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventDestroy failed !\n");
		return cudaStatus;
	}

	cudaStatus = hipEventDestroy(*stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventDestroy failed !\n");
		return cudaStatus;
	}

	return hipSuccess;
}

int main()
{
    float a[SIZE*SIZE];
    float b[SIZE*SIZE];
    float c[SIZE*SIZE];
	float msecTotal = 0.0f;
	for (int i = 0; i < SIZE*SIZE; ++i)
	{
		a[i] = 1.0;// (float)rand() / RAND_MAX;;
		b[i] = 2.0;// (float)rand() / RAND_MAX;;
	}

	printMatrix(a, SIZE);
	printf("*\n");
	printMatrix(b, SIZE);
	
    // Add vectors in parallel.

	hipError_t cudaStatus = multiplyMatrix((float*)a, (float*)b, (float*)c, SIZE, msecTotal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyMatrix failed!");
        return 1;
    }

	printf("=\n");
	printMatrix(c, SIZE);

	printf("\nTotal miliseconds: %.5f\n", msecTotal);

	if (0 < cudaCall(hipDeviceReset, "hipDeviceReset failed!")) return 1;

    return 0;
}

#define CUDA_CHECK(FUN)				\
{									\
	hipError_t status;				\
	status = FUN;					\
	if (status != hipSuccess)		\
		return status;				\
}
		

hipError_t multiplyMatrix(float *a, float *b, float *c, const int size, float & msecTotal)
{
	const size_t totalSize = size * size * sizeof(float);

	auto cudaMemoryDeleter = [&](float* ptr) { hipFree(ptr); };
	
	std::shared_ptr<float> dev_a(new float(), cudaMemoryDeleter);
	std::shared_ptr<float> dev_b(new float(), cudaMemoryDeleter);
	std::shared_ptr<float> dev_c(new float(), cudaMemoryDeleter);

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	CUDA_CHECK(cudaAllocate((void**)&dev_a, totalSize));
	CUDA_CHECK(cudaAllocate((void**)&dev_b, totalSize));
	CUDA_CHECK(cudaAllocate((void**)&dev_c, totalSize));

	CUDA_CHECK(cudaCopy(dev_a.get(), a, totalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(cudaCopy(dev_b.get(), b, totalSize, hipMemcpyHostToDevice));

	dim3 blockDim(size, size);
	hipEvent_t start, stop;

	CUDA_CHECK(cudaStartTimer(&start, &stop));

	multiplyKernet <<< 1, blockDim, size*size >>> (dev_a.get(), dev_b.get(), dev_c.get());

	CUDA_CHECK(cudaStopTimer(&start, &stop, msecTotal));

	CUDA_CHECK(cudaCall(hipGetLastError, "addKernet lauch failed"));
	CUDA_CHECK(cudaCall(hipDeviceSynchronize, "hipDeviceSynchronize returned error code"));
	CUDA_CHECK(cudaCopy(c, dev_c.get(), totalSize, hipMemcpyDeviceToHost));
	
	return hipSuccess;
}

void printMatrix(float *arr, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			printf("%.5f, ", arr[i*size + j]);
		}
		printf("\n");
	}
}
