#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void multiplyKernet(float *a, float *b, float *c, int size)
{
	extern __shared__ float t_c[];
	int i = threadIdx.x;
	int j = threadIdx.y;
	float loc_c = 0.0;
	__syncthreads();
	for (int k = 0; k < size; ++k)
	{
		loc_c += a[i*k+j] * b[j*k+i];
	}
	c[i*size + j] = loc_c;
	__syncthreads();
}


int cudaCopy(float *dest, float *src, size_t size, hipMemcpyKind copyKind)
{
	hipError_t cudaStatus = hipMemcpy(dest, src, size, copyKind);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed: %s", hipGetErrorString(cudaStatus));
		return -1;
	}
	return 0;
}

int cudaAllocate(void **devPtr, size_t size)
{
	hipError_t cudaStatus = hipMalloc(devPtr, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed: %s", hipGetErrorString(cudaStatus));
		return -1;
	}
	return 0;
}

int cudaCall(hipError_t(*cudaFunc)(), char *msg)
{
	hipError_t cudaStatus = cudaFunc();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(cudaStatus));
		return -1;
	}
	return 0;
}

int cudaStartTimer(hipEvent_t *start, hipEvent_t *stop)
{
	hipError_t cudaStatus = hipEventCreate(start);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudaCEventCreate(start) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	cudaStatus = hipEventCreate(stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventCreate(stop) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	cudaStatus = hipEventRecord(*start, 0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventRecord(start) failed (errorCode: %s)!\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	return 0;
}

int cudaStopTimer(hipEvent_t *start, hipEvent_t *stop, float *msecTotal)
{
	hipError_t cudaStatus = hipEventRecord(*stop, 0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventRecord(stop) failed !\n");
		return -1;
	}
	cudaStatus = hipEventSynchronize(*stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventSynchronize failed !\n");
		return -1;
	}
	*msecTotal = 0.0f;
	cudaStatus = hipEventElapsedTime(msecTotal, *start, *stop);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipEventElapsedTime failed !\n");
		return -1;
	}
	hipEventDestroy(*start);
	hipEventDestroy(*stop);
	return 0;
}

int main()
{
    float a[SIZE*SIZE];
    float b[SIZE*SIZE];
    float c[SIZE*SIZE];
	float msecTotal = 0.0f;
	for (int i = 0; i < SIZE*SIZE; ++i)
	{
		a[i] = 1.0;// (float)rand() / RAND_MAX;;
		b[i] = 2.0;// (float)rand() / RAND_MAX;;
	}

	printMatrix(a, SIZE);
	printf("*\n");
	printMatrix(b, SIZE);
	
    // Add vectors in parallel.

	hipError_t cudaStatus = multiplyMatrix((float*)a, (float*)b, (float*)c, SIZE, &msecTotal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyMatrix failed!");
        return 1;
    }

	printf("=\n");
	printMatrix(c, SIZE);

	printf("\nTotal miliseconds: %.5f\n", msecTotal);

	if (0 < cudaCall(hipDeviceReset, "hipDeviceReset failed!")) return 1;
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

hipError_t multiplyMatrix(float *a, float *b, float *c, const int size, float *msecTotal)
{
	float *dev_a;
	float *dev_b;
	float *dev_c;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	if (0 > cudaAllocate((void**)&dev_a, size*size * sizeof(float))) goto Error;
	if (0 > cudaAllocate((void**)&dev_b, size*size * sizeof(float))) goto Error;
	if (0 > cudaAllocate((void**)&dev_c, size*size * sizeof(float))) goto Error;

	if (0 > cudaCopy(dev_a, a, size*size * sizeof(float), hipMemcpyHostToDevice)) goto Error;
	if (0 > cudaCopy(dev_b, b, size*size * sizeof(float), hipMemcpyHostToDevice)) goto Error;
		
	dim3 blockDim(size, size);
	hipEvent_t start, stop;

	if(0 > cudaStartTimer(&start, &stop)) goto Error;

	multiplyKernet <<<1, blockDim, size*size >>> (dev_a, dev_b, dev_c, size);

	if(0 > cudaStopTimer(&start, &stop, msecTotal)) goto Error;
	
	if (0 > cudaCall(hipGetLastError, "addKernet lauch failed") ) goto Error;
	if (0 > cudaCall(hipDeviceSynchronize, "hipDeviceSynchronize returned error code")) goto Error;
	if (0 > cudaCopy(c, dev_c, size * size * sizeof(float), hipMemcpyDeviceToHost)) goto Error;
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching multiplyKernet!\n", cudaStatus);

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return cudaStatus;
}

void printMatrix(float *arr, const int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			printf("%.5f, ", arr[i*size + j]);
		}
		printf("\n");
	}
}
